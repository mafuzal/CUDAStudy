//nvcc -arch sm_61 hello.cu -o hello 


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (void)
{
   printf("Hello World from GPU and from thread #:%d.!\n", threadIdx.x);
}
int main(void)
{
   //hello from cpu
   printf("Hello World from CPU!\n");
   helloFromGPU <<<1, 10>>>();
   hipDeviceReset();
   return 0;
}
